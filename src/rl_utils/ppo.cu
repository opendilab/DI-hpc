#include "hpc/rll/cuda/rl_utils/entry.h"
#include "hpc/rll/cuda/rl_utils/ppo_kernel.h"

namespace hpc {
namespace rll {
namespace cuda {

void PPOForward(
    const std::vector<torch::Tensor>& inputs,
    std::vector<torch::Tensor>& outputs,
    bool use_value_clip,
    float clip_ratio,
    float dual_clip) {

    unsigned int index = 0;
    const torch::Tensor& logits_new = inputs[index++];
    const torch::Tensor& logits_old = inputs[index++];
    const torch::Tensor& action = inputs[index++];
    const torch::Tensor& value_new = inputs[index++];
    const torch::Tensor& value_old = inputs[index++];
    const torch::Tensor& adv = inputs[index++];
    const torch::Tensor& return_ = inputs[index++];
    const torch::Tensor& weight = inputs[index++];

    index = 0;
    torch::Tensor& logits_new_prob = outputs[index++];
    torch::Tensor& logits_new_entropy = outputs[index++];
    torch::Tensor& logits_new_grad_logits = outputs[index++];
    torch::Tensor& logits_new_grad_prob = outputs[index++];
    torch::Tensor& logits_new_grad_entropy = outputs[index++];
    torch::Tensor& logits_old_prob = outputs[index++];
    torch::Tensor& grad_policy_loss_buf = outputs[index++];
    torch::Tensor& grad_value_loss_buf = outputs[index++];
    torch::Tensor& grad_entropy_loss_buf = outputs[index++];
    torch::Tensor& policy_loss = outputs[index++];
    torch::Tensor& value_loss = outputs[index++];
    torch::Tensor& entropy_loss = outputs[index++];
    torch::Tensor& approx_kl = outputs[index++];
    torch::Tensor& clipfrac = outputs[index++];

    checkCudaErr(hipMemsetAsync(policy_loss.data_ptr<float>(), 0, sizeof(float)));
    checkCudaErr(hipMemsetAsync(value_loss.data_ptr<float>(), 0, sizeof(float)));
    checkCudaErr(hipMemsetAsync(entropy_loss.data_ptr<float>(), 0, sizeof(float)));
    checkCudaErr(hipMemsetAsync(approx_kl.data_ptr<float>(), 0, sizeof(float)));
    checkCudaErr(hipMemsetAsync(clipfrac.data_ptr<float>(), 0, sizeof(float)));

    const unsigned int batch_size = logits_new.size(0);
    const unsigned int num_output = logits_new.size(1);
    {
        unsigned int block_size = DEFAULT_WARP_NUM * WARP_SIZE;
        unsigned int grid_size = batch_size;
        categoricalProbEntropy<<<grid_size, block_size>>>(
                num_output, logits_new.data_ptr<float>(), action.data_ptr<int64_t>(),
                logits_new_prob.data_ptr<float>(), logits_new_entropy.data_ptr<float>(),
                logits_new_grad_logits.data_ptr<float>(), logits_new_grad_prob.data_ptr<float>(),
                logits_new_grad_entropy.data_ptr<float>());
        categoricalProb<<<grid_size, block_size>>>(
                num_output, logits_old.data_ptr<float>(), action.data_ptr<int64_t>(), logits_old_prob.data_ptr<float>());
    }
    {
        unsigned int block_size = DEFAULT_WARP_NUM * WARP_SIZE;
        unsigned int grid_size = (batch_size + block_size - 1) / block_size;
        ppoLoss<<<grid_size, block_size>>>(
                batch_size, value_new.data_ptr<float>(), value_old.data_ptr<float>(),
                logits_new_prob.data_ptr<float>(), logits_old_prob.data_ptr<float>(), logits_new_entropy.data_ptr<float>(),
                adv.data_ptr<float>(), return_.data_ptr<float>(), weight.data_ptr<float>(),
                use_value_clip, clip_ratio, dual_clip,
                policy_loss.data_ptr<float>(), value_loss.data_ptr<float>(), entropy_loss.data_ptr<float>(),
                approx_kl.data_ptr<float>(), clipfrac.data_ptr<float>(),
                grad_policy_loss_buf.data_ptr<float>(), grad_value_loss_buf.data_ptr<float>(), grad_entropy_loss_buf.data_ptr<float>());
    }
}

void PPOBackward(
    const std::vector<torch::Tensor>& inputs,
    std::vector<torch::Tensor>& outputs) {

    unsigned int index = 0;
    const torch::Tensor& grad_policy_loss = inputs[index++];
    const torch::Tensor& grad_value_loss = inputs[index++];
    const torch::Tensor& grad_entropy_loss = inputs[index++];
    const torch::Tensor& grad_policy_loss_buf = inputs[index++];
    const torch::Tensor& grad_value_loss_buf = inputs[index++];
    const torch::Tensor& grad_entropy_loss_buf = inputs[index++];
    const torch::Tensor& logits_new_grad_logits = inputs[index++];
    const torch::Tensor& logits_new_grad_prob = inputs[index++];
    const torch::Tensor& logits_new_grad_entropy = inputs[index++];

    index = 0;
    torch::Tensor& grad_value = outputs[index++];
    torch::Tensor& grad_logits_new = outputs[index++];

    const unsigned int batch_size = grad_logits_new.size(0);
    const unsigned int num_output = grad_logits_new.size(1);
    {
        unsigned int block_size = DEFAULT_WARP_NUM * WARP_SIZE;
        unsigned int grid_size = (batch_size + block_size - 1) / block_size;
        ppoBackwardValueNew<<<grid_size, block_size>>>(
                batch_size, grad_value_loss.data_ptr<float>(), grad_value_loss_buf.data_ptr<float>(), grad_value.data_ptr<float>());
    }
    {
        unsigned int block_size = DEFAULT_WARP_NUM * WARP_SIZE;
        unsigned int grid_size = batch_size;
        ppoBackwardLogitsNew<<<grid_size, block_size>>>(
                batch_size, num_output, grad_policy_loss.data_ptr<float>(), grad_entropy_loss.data_ptr<float>(),
                grad_policy_loss_buf.data_ptr<float>(), grad_entropy_loss_buf.data_ptr<float>(),
                logits_new_grad_logits.data_ptr<float>(), logits_new_grad_prob.data_ptr<float>(), logits_new_grad_entropy.data_ptr<float>(),
                grad_logits_new.data_ptr<float>());
    }
}

}  // namespace cuda
}  // namespace rll
}  // namespace hpc

